#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <ctime>
//#include <Eigen/dense>
#include "Vector3f.h"

struct Vector3i
{
	int x;
	int y;
	int z;
};

struct TriangleMesh
{
	int numOfVerts = 0;
	int numOfTris = 0;
	Vector3f* verts = nullptr;
	Vector3i* tris = nullptr;
};

static void parseOBJFile(const std::string filePath, TriangleMesh* triMesh)
{
	std::ifstream stream(filePath);

	std::string line;

	int vert_iterator = 0;
	int tris_iterator = 0;

	while (getline(stream, line, ' '))
	{
		if (line == "v") //v x y z
		{
			Vector3f vertex;
			getline(stream, line, ' ');
			vertex.x = std::stof(line);
			getline(stream, line, ' ');
			vertex.y = std::stof(line);
			getline(stream, line, '\n');
			vertex.z = std::stof(line);

			*(triMesh->verts + vert_iterator++) = vertex;
			continue;
		}
		if (line == "f") //f  v1/vt1/vn1 ..
		{
			Vector3i face;
			getline(stream, line, '/');
			face.x = std::stoi(line);
			getline(stream, line, ' ');
			getline(stream, line, '/');
			face.y = std::stoi(line);
			getline(stream, line, ' ');
			getline(stream, line, '/');
			face.z = std::stoi(line);

			*(triMesh->tris + tris_iterator++) = face;
		}
		getline(stream, line, '\n');
	}

	triMesh->numOfVerts = vert_iterator;
	triMesh->numOfTris = tris_iterator;


	stream.close();
}

struct pixel
{
	float r;
	float g;
	float b;
};

__global__ void shadePixels(pixel* p)
{
	int pixel_x = threadIdx.x;
	int pixel_y = blockIdx.x;

	//pixels are stored colunm/block num by row/thread offset
	//-----------------------------
	//---------------x-------------
	//-----------------------------
	p = p + (pixel_y * blockDim.x) + pixel_x;

	//set the film size
	const float image_plane_width = 0.07f; // 7cm
	const float image_plane_height = 0.06f; //6cm

	//calculate pixel area to sample over (coordinates begin at the top left of each pixel)
	const float pixel_width = image_plane_width / blockDim.x;

	//this is where pixel sample points should be generated, before conversion to camera space

	//find this pixel's camera space position
	Vector3f pixelpos;
	pixelpos.x = (float) pixel_x / blockDim.x;
	pixelpos.x = pixelpos.x * image_plane_width - (image_plane_width / 2.0f);
	pixelpos.y = (float) pixel_y / gridDim.x;
	pixelpos.y = -pixelpos.y * image_plane_height + (image_plane_height / 2.0f);
	pixelpos.z = 0.f;

	//select a focal length, aperature, and focus distance
	const float focal_length = 0.05f;
	const float aperture = 16.0f;
	const float focus_dist = 3.f;

	//Set V and W ... TODO add lens configurations
	Vector3f V = Vector3f(0.f, 0.f, focal_length);
	Vector3f W = Vector3f(V.x + pixelpos.x,  V.y + pixelpos.y, -1.f * V.z);

	//set the background color
	Vector3f color = Vector3f(0.2f, 0.3f, 0.4f);

	// TEMP DEFINE SPHERE DATA
	sph

	//Set the final color
	p->r *= 255;
	p->g *= 255;
	p->b *= 255;

}

int main()
{
	const int image_width = 1024;
	const int image_height = 878;

	//create the pixel array
	pixel* pixels;

	hipMallocManaged(&pixels, sizeof(pixel) * image_width * image_height);

	/*
	//Read in Triangle information from OBJ file
	TriangleMesh* trimesh;
	hipMallocManaged(&trimesh, sizeof(TriangleMesh));
	hipMallocManaged(&trimesh->verts, sizeof(Vector3f) * 1000);
	hipMallocManaged(&trimesh->tris, sizeof(Vector3i) * 1000);

	parseOBJFile("in/mnky.obj", trimesh);
	*/

	//Create lights, for now a light is a point

	//Create spheres, a point and a radius

	auto start_p1 = std::chrono::system_clock::now();

	//run the kernel, <<< numOf Blocks, numOf threads per block >>>
	shadePixels <<<image_height, image_width>>> (pixels);

	hipDeviceSynchronize();

	auto start_p2 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p1 = start_p2 - start_p1;

	//save the output to a file
	std::ofstream image_of;
	image_of.open("out/img.ppm", std::ios::out);

	if (!image_of.is_open())
	{
		std::cout << "File does not exist";
	}

	image_of << "P3\n" << image_width << ' ' << image_height << "\n255\n";

	for (int i = 0; i < image_height; i++)
	{ 
		for (int k = 0; k < image_width; k++)
		{
			pixel* p = pixels + (i * image_width) + k;
			
			image_of << static_cast<int>(p->r) << ' ' << static_cast<int>(p->g) << ' ' << static_cast<int>(p->b) << ' ';
		}
		image_of << "\n";
	}

	auto start_p3 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p2 = start_p3 - start_p2;

	std::cout << "GPU computation elapsed time: " << elapsed_p1.count() << "s\n";
	std::cout << "Image output elapsed time: " << elapsed_p2.count() << "s\n";

	//clean up
	image_of.close();
	hipFree(trimesh->verts);
	hipFree(trimesh->tris);
	hipFree(trimesh);
	hipFree(pixels);
	hipDeviceReset();

	return 0;
}