#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <ctime>
#include <algorithm>
#include "GLM/glm/vec3.hpp"
#include "GLM/glm/vec4.hpp"
#include "GLM/glm/geometric.hpp"
//#include "GLM/glm/common.hpp"

struct Vector4i
{
	int x;
	int y;
	int z;
	int vn1;
};

struct TriangleMesh
{
	int numOfVerts = 0;
	int numOfTris = 0;
	glm::vec3* verts = nullptr;
	glm::vec3* vnorms = nullptr;
	Vector4i* tris = nullptr;
};

static void parseOBJFile(const std::string filePath, TriangleMesh* triMesh)
{
	std::ifstream stream(filePath);

	std::string line;

	int vert_iterator = 0;
	int tris_iterator = 0;
	int vnorm_iterator = 0;

	while (getline(stream, line, ' '))
	{
		if (line == "v") //v x y z
		{
			glm::vec3 vertex;
			getline(stream, line, ' ');
			vertex.x = std::stof(line);
			getline(stream, line, ' ');
			vertex.y = std::stof(line);
			getline(stream, line, '\n');
			vertex.z = std::stof(line);

			*(triMesh->verts + vert_iterator++) = vertex;
			continue;
		}
		if (line.compare("vn") == 0)
		{
			glm::vec3 vnorm;
			getline(stream, line, ' ');
			vnorm.x = std::stof(line);
			getline(stream, line, ' ');
			vnorm.y = std::stof(line);
			getline(stream, line, '\n');
			vnorm.z = std::stof(line);

			*(triMesh->vnorms + vnorm_iterator++) = vnorm;
			continue;
		}
		if (line == "f") //f  v1/vt1/vn1 ..
		{
			Vector4i face;
			getline(stream, line, '/');
			face.x = std::stoi(line);
			getline(stream, line, '/');
			face.vn1 = std::stoi(line);
			getline(stream, line, ' ');
			getline(stream, line, '/');
			face.y = std::stoi(line);
			getline(stream, line, ' ');
			getline(stream, line, '/');
			face.z = std::stoi(line);

			*(triMesh->tris + tris_iterator++) = face;
		}
		getline(stream, line, '\n');
	}

	triMesh->numOfVerts = vert_iterator;
	triMesh->numOfTris = tris_iterator;


	stream.close();
}

struct sphere
{
	glm::vec3 location;
	float r;

	__device__ sphere(float x, float y, float z, float r)
	{
		location.x = x;
		location.y = y;
		location.z = z;
		this->r = r;
	};
};

struct pointLight
{
	glm::vec3 location;
	glm::vec3 color;

	float i; //intensity

	__device__ pointLight(float x, float y, float z, float r, float g, float b, float intensity)
	{
		location.x = x;
		location.y = y;
		location.z = z;
		color.x = r;
		color.y = g;
		color.z = b;
		i = intensity;
	};
};

struct material
{
	glm::vec3 diffuse;
	glm::vec3 ambient;
	glm::vec3 specular;
	float power;

	__device__ material(float r1, float g1, float b1, float r2, float g2, float b2, float r3, float g3, float b3, float p)
	{
		diffuse.r = r1;
		diffuse.g = g1;
		diffuse.b = b1;
		ambient.r = r2;
		ambient.g = g2;
		ambient.b = b2;
		specular.r = r3;
		specular.g = g3;
		specular.b = b3;
		power = p;
	};
};
/*
__device__ inline float raySphere(glm::vec3 V, glm::vec3 W, sphere S) {
	V = V - S.location + W * 0.001f;
	float b = dot(V, W);
	float d = b * b - dot(V, V) + S.r * S.r;
	return d < 0. ? -1. : -b - sqrt(d);
}*/

__device__ inline float rayHalfspace(glm::vec3 V, glm::vec3 W, glm::vec4 H) {
	glm::vec4 V1 = glm::vec4(V, 1.);
	glm::vec4 W0 = glm::vec4(W, 0.);
	return -dot(V1, H) / dot(W0, H);
}

__device__ glm::vec3 shadePoint(glm::vec3 P, glm::vec3 W, glm::vec3 N, material* m, pointLight* lights, int num_lights)
{
	glm::vec3 c = m->ambient;
	for (int l = 0; l < num_lights; l++) {

		//calculate the light direction from the point lamps
		glm::vec3 Ld = (lights + l)->location - P;
		float distance = length(Ld);
		distance = distance * distance;
		Ld = normalize(Ld);

		// TODO: shadows for all surfaces
		glm::vec3 Rd = 2.f * dot(N, Ld) * N - Ld;
		c += (lights + l)->location *
			m->diffuse * glm::max(0.f, dot(N, Ld));
		c += m->specular * pow(glm::max(0.f, dot(Rd, -W)), m->power);
		c *= (lights + l)->location / distance;
	}

	return c;
}

/*
__device__ inline glm::vec3 shadeSphere(glm::vec3 P, glm::vec3 W, glm::vec3 N, sphere* spheres, int num_spheres, material m, pointLight* lights, int num_lights) 
{
	glm::vec3 c = m.ambient;

	for (int l = 0; l < num_lights; l++) {

		// SPHERE SHADOWS

		float t = -1.;
		for (int n = 0; n < num_spheres; n++)
			t = glm::max(t, raySphere(P, (lights+n)->location, *(spheres+n)));

		// IF NOT, ADD LIGHTING FROM THIS LIGHT

		if (t < 0.) 
		{
			glm::vec3 R = 2.f * dot(N, (lights + l)->location) * N - (lights + l)->location;
			c += (lights + l)->location *
				m.diffuse * glm::max(0.f, dot(N, (lights + l)->location));
			c += m.specular * pow(glm::max(0.f, dot(R, -W)), m.power);
		}
	}

	return c;
}*/

__device__ glm::vec3 triangleTest(glm::vec3 V, glm::vec3 W, glm::vec3 a, glm::vec3 b, glm::vec3 c, glm::vec3 n, material* m, pointLight* light, int num_lights)
{
	glm::vec3 ba = a - b;
	glm::vec3 bc = c - b;
	//glm::vec3 bPerp = cross(bc, ba); //plane normal
	//glm::vec3 n = normalize(bPerp);
	float d = -dot(n, a);
	glm::vec4 H = glm::vec4(n.x, n.y, n.z, d);

	//find t
	float t = rayHalfspace(V, W, H);

	//if t is positive we hit the plane infront of the camera
	if (t > 0. && t < 1000.)
	{
		//calculate point p
		glm::vec3 P = V + t * W;

		//make sure p is within the edges
		glm::vec3 ab = b - a;
		glm::vec3 ap = P - a;
		if (dot(cross(ab, ap), n) < 0.) // > 0 means inside, = 0 means on the edge
		{
			return glm::vec3(0.);
		}

		glm::vec3 bp = P - b;
		if (dot(cross(bc, bp), n) < 0.)
		{
			return glm::vec3(0.);
		}

		glm::vec3 ca = a - c;
		glm::vec3 cp = P - c;
		if (dot(cross(ca, cp), n) < 0.)
		{
			return glm::vec3(0.);
		}

		//TODO calculate the barycentric coordinates to apply smooth shading with vertex normals

		return shadePoint(P, W, n, m, light, num_lights);		
	}
	return glm::vec3(0.);
}

/*Each vector in indicies contains the three points that make a triangle,
points are indexes of vertex coordinates stored in the verticies array*/
__device__ inline glm::vec3 drawTriangles(glm::vec3 V, glm::vec3 W, TriangleMesh* trimesh, material* m, pointLight* lights, int num_lights)
{
	//use indicies to draw triangles with vertex array
	for (int i = 0; i < trimesh->numOfTris; i++)
	{
		//vertex indexes begin at 1 for obj files
		glm::vec3 color = triangleTest(V, W, *(trimesh->verts + (trimesh->tris + i)->x - 1), *(trimesh->verts + (trimesh->tris + i)->y - 1),
			*(trimesh->verts + (trimesh->tris + i)->z - 1), *(trimesh->vnorms + (trimesh->tris + i)->vn1 - 1), m, lights, num_lights);

		if (dot(color, glm::vec3(1.f)) > 0.)
		{
			return color;
		}
	}

	return glm::vec3(0.);
}

/*
__device__ inline glm::vec3 raySpheres(float tMin, glm::vec3 V, glm::vec3 W, sphere* spheres, int num_spheres, glm::vec3 color, material m, pointLight* lights, int num_lights)
{
	for (int n = 0; n < num_spheres; n++)
	{
		float t = raySphere(V, W, *(spheres + n));
		if (t > 0. && t < tMin) {
			glm::vec3 P = V + (W * t);
			glm::vec3 N = normalize(P - (spheres + n)->location);
			color = shadeSurface(P, W, N, spheres, num_spheres, m, lights, num_lights);
			tMin = t;
		}
	}
	return color;
}*/

__global__ void shadePixels(glm::vec3* p, TriangleMesh* trimesh)
{
	int pixel_x = threadIdx.x;
	int pixel_y = blockIdx.x;

	//pixels are stored colunm/block num by row/thread offset
	//-----------------------------
	//---------------x-------------
	//-----------------------------
	p = p + (pixel_y * blockDim.x) + pixel_x;

	//set the film size
	const float image_plane_width = 0.07f; // 7cm
	const float image_plane_height = 0.06f; //6cm

	//calculate pixel area to sample over (coordinates begin at the top left of each pixel)
	//const float pixel_width = image_plane_width / blockDim.x;

	//this is where pixel sample points should be generated, before conversion to camera space

	//find this pixel's camera space position
	glm::vec3 pixelpos;
	pixelpos.x = (float) pixel_x / blockDim.x;
	pixelpos.x = pixelpos.x * image_plane_width - (image_plane_width / 2.0f);
	pixelpos.y = (float) pixel_y / gridDim.x;
	pixelpos.y = -pixelpos.y * image_plane_height + (image_plane_height / 2.0f);
	pixelpos.z = 0.f;

	//select a focal length, aperature, and focus distance
	const float focal_length = 0.05f;
	//const float aperture = 16.0f;
	//const float focus_dist = 3.f;

	//Set V and W ... TODO add lens configurations
	glm::vec3 V = glm::vec3(0.f, 0.f, focal_length);
	glm::vec3 W = glm::vec3(V.x + pixelpos.x,  V.y + pixelpos.y, -1.f * V.z);

	//set the background color
	glm::vec3 color = glm::vec3(0.2f, 0.3f, 0.4f);

	// TEMP DEFINE SPHERE DATA
	//sphere spheres[3] = {sphere(0.f, 0.f, -5.f, 1.f), sphere(0.5f, 0.f, -3.f, 1.f), sphere(-0.5f, 0.f, -7.f, 1.f)};
	// TEMP DEFINE LIGHT DATA
	pointLight light = pointLight(0.f, 1.f, -.5f, .59f, .93f, .59f, 40.f);
	// TEMP DEFINE MATERIAL DATA
	material sphere_mat = material(0.1f, 0.1f, 0.1f, 0.2f, 0.4f, 0.5f, 1.f, 1.f, 1.f, 1.5f);

	//color = raySpheres(-1.f, V, W, &spheres[0], 3, color, sphere_mat, &lights[0], 1);

	color += drawTriangles(V, W, trimesh, &sphere_mat, &light, 1);
	//color += triangleTest(V, W, *(trimesh->verts + (trimesh->tris)->x), *(trimesh->verts + (trimesh->tris)->y),
		//*(trimesh->verts + (trimesh->tris)->z), &sphere_mat, &light, 1);
	//color += triangleTest(V, W, glm::vec3(-0.1f, 0.0f, -1.f), glm::vec3(0.1f, 0.0f, -1.f), glm::vec3(0.f, 0.1f, -1.f), &sphere_mat, &light, 1);

	//Set the final color
	p->r = color.x * 255;
	p->g = color.y * 255;
	p->b = color.z * 255;

}

int main()
{
	const int image_width = 1024;
	const int image_height = 878;

	//create the pixel array
	glm::vec3* pixels;

	hipMallocManaged(&pixels, sizeof(glm::vec3) * image_width * image_height);

	
	//Read in Triangle information from OBJ file
	TriangleMesh* trimesh;
	glm::vec3* verts;
	glm::vec3* vnorms;
	Vector4i* tris;
	hipMallocManaged(&trimesh, sizeof(TriangleMesh));
	hipMallocManaged(&verts, sizeof(glm::vec3) * 1000);
	hipMallocManaged(&vnorms, sizeof(glm::vec3) * 1000);
	hipMallocManaged(&tris, sizeof(Vector4i) * 1000);
	trimesh->verts = verts;
	trimesh->vnorms = vnorms;
	trimesh->tris = tris;

	auto start_p0 = std::chrono::system_clock::now();

	parseOBJFile("in/mnky.obj", trimesh);

	auto start_p1 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p0 = start_p0 - start_p1;

	//run the kernel, <<< numOf Blocks, numOf threads per block >>>
	shadePixels <<<image_height, image_width>>> (pixels, trimesh);

	hipDeviceSynchronize();

	auto start_p2 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p1 = start_p2 - start_p1;

	//save the output to a file
	std::ofstream image_of;
	image_of.open("out/img.ppm", std::ios::out);

	if (!image_of.is_open())
	{
		std::cout << "File does not exist";
	}

	image_of << "P3\n" << image_width << ' ' << image_height << "\n255\n";

	for (int i = 0; i < image_height; i++)
	{ 
		for (int k = 0; k < image_width; k++)
		{
			glm::vec3* p = pixels + (i * image_width) + k;
			
			image_of << static_cast<int>(p->r) << ' ' << static_cast<int>(p->g) << ' ' << static_cast<int>(p->b) << ' ';
		}
		image_of << "\n";
	}

	auto start_p3 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p2 = start_p3 - start_p2;

	std::cout << "Obj input elapsed time: " << elapsed_p0.count() << "s\n";
	std::cout << "GPU computation elapsed time: " << elapsed_p1.count() << "s\n";
	std::cout << "Image output elapsed time: " << elapsed_p2.count() << "s\n";

	//clean up
	image_of.close();
	hipFree(trimesh->verts);
	hipFree(trimesh->vnorms);
	hipFree(trimesh->tris);
	hipFree(trimesh);
	hipFree(pixels);
	hipDeviceReset();

	return 0;
}