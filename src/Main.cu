#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <ctime>
#include <algorithm>
#include "GLM/glm/vec3.hpp"
#include "GLM/glm/vec4.hpp"
#include "GLM/glm/geometric.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
//#include "SimplexNoise.h"

struct face
{
	int x;
	int y;
	int z;
	int vn1;
	int vn2;
	int vn3;
};

struct material
{
	glm::vec3 diffuse;
	glm::vec3 ambient;
	glm::vec3 specular;
	float power;

	__device__ material(float r1, float g1, float b1, float r2, float g2, float b2, float r3, float g3, float b3, float p)
	{
		diffuse.r = r1;
		diffuse.g = g1;
		diffuse.b = b1;
		ambient.r = r2;
		ambient.g = g2;
		ambient.b = b2;
		specular.r = r3;
		specular.g = g3;
		specular.b = b3;
		power = p;
	};
};

struct VertexData
{
	int numOfVerts = 0;
	glm::vec3* verts = nullptr;
	glm::vec3* vnorms = nullptr;
	//TODO: add vertex textures
};

struct TriangleMesh
{
	char name[10];
	material* mat;
	int numOfTris = 0;
	face* tris = nullptr;
};

static void parseOBJFile(const std::string filePath, VertexData* vertData, TriangleMesh* triMeshes, int maxMeshes, int &numOfMeshes)
{
	std::ifstream stream(filePath);

	std::string line;

	int mesh_iterator = -1; //gets set to 0 on first obj scan
	int vert_iterator = 0;
	int tris_iterator = 0;
	int vnorm_iterator = 0;
	//int vert_culmlative = 0;
	//int vn_culmlative = 0;

	while (getline(stream, line, ' '))
	{
		if (line == "v") //v x y z
		{
			glm::vec3 vertex;
			getline(stream, line, ' ');
			vertex.x = std::stof(line);
			getline(stream, line, ' ');
			vertex.y = std::stof(line);
			getline(stream, line, '\n');
			vertex.z = std::stof(line);

			*(vertData->verts + vert_iterator++) = vertex;
			continue;
		}
		if (line.compare("vn") == 0)
		{
			glm::vec3 vnorm;
			getline(stream, line, ' ');
			vnorm.x = std::stof(line);
			getline(stream, line, ' ');
			vnorm.y = std::stof(line);
			getline(stream, line, '\n');
			vnorm.z = std::stof(line);

			*(vertData->vnorms + vnorm_iterator++) = vnorm;
			continue;
		}
		if (line == "f") //f  v1/vt1/vn1 ..
		{
			face face;
			getline(stream, line, '/');
			face.x = std::stoi(line);
			getline(stream, line, '/');
			getline(stream, line, ' ');
			face.vn1 = std::stoi(line);
			getline(stream, line, '/');
			face.y = std::stoi(line);
			getline(stream, line, '/');
			getline(stream, line, ' ');
			face.vn2 = std::stoi(line);
			getline(stream, line, '/');
			face.z = std::stoi(line);
			getline(stream, line, '/');
			getline(stream, line, '\n');
			face.vn3 = std::stoi(line);

			*((triMeshes + mesh_iterator)->tris + tris_iterator++) = face;
			(triMeshes + mesh_iterator)->numOfTris = tris_iterator;
			continue;
		}
		if (line == "o") //object name
		{
			if (mesh_iterator == maxMeshes-1)
			{
				break; //max number of meshes reached
			}

			//prepare scanning for next mesh
			mesh_iterator++;
			tris_iterator = 0;

			getline(stream, line, '\n');
			std::strcpy((triMeshes + mesh_iterator)->name, line.c_str());		

			continue;
		}
		getline(stream, line, '\n');
	}
	vertData->numOfVerts = vert_iterator;
	numOfMeshes = mesh_iterator+1;
	stream.close();
}

struct sphere
{
	glm::vec3 location;
	float r;

	__device__ sphere(float x, float y, float z, float r)
	{
		location.x = x;
		location.y = y;
		location.z = z;
		this->r = r;
	};
};

struct pointLight
{
	glm::vec3 location;
	glm::vec3 color;

	float i; //intensity

	__device__ pointLight(float x, float y, float z, float r, float g, float b, float intensity)
	{
		location.x = x;
		location.y = y;
		location.z = z;
		color.x = r;
		color.y = g;
		color.z = b;
		i = intensity;
	};
};

/*
__device__ inline float raySphere(glm::vec3 V, glm::vec3 W, sphere S) {
	V = V - S.location + W * 0.001f;
	float b = dot(V, W);
	float d = b * b - dot(V, V) + S.r * S.r;
	return d < 0. ? -1. : -b - sqrt(d);
}*/

__device__ inline float rayHalfspace(glm::vec3 V, glm::vec3 W, glm::vec4 H) {
	glm::vec4 V1 = glm::vec4(V, 1.);
	glm::vec4 W0 = glm::vec4(W, 0.);
	return -dot(V1, H) / dot(W0, H);
}

/* Return a point P if V, W intersects triangle ABC */
__device__ inline bool triangleTest(glm::vec3 V, glm::vec3 W, glm::vec3 a, glm::vec3 b, glm::vec3 c, glm::vec3 N, glm::vec3 &P)
{
	float d = -dot(N, a);
	glm::vec4 H = glm::vec4(N.x, N.y, N.z, d);

	//find t
	float t = rayHalfspace(V, W, H);

	//if t is positive we hit the plane infront of the camera
	if (t > 0.f && t < 1000.f)
	{
		//calculate point p
		glm::vec3 p = V + t * W;

		//make sure p is within the edges
		glm::vec3 ab = b - a;
		glm::vec3 ap = p - a;
		if (dot(cross(ab, ap), N) < 0.f) // > 0 means inside, = 0 means on the edge
		{
			return false;
		}

		glm::vec3 bc = c - b;
		glm::vec3 bp = p - b;
		if (dot(cross(bc, bp), N) < 0.f)
		{
			return false;
		}

		glm::vec3 ca = a - c;
		glm::vec3 cp = p - c;
		if (dot(cross(ca, cp), N) < 0.f)
		{
			return false;
		}

		//TODO calculate the barycentric coordinates to apply smooth shading with vertex normals

		//set values
		P = p;
		return true;
	}
	return false;
}

/*Returns a point P of the closest triangle*/
__device__ inline bool testTriangles(glm::vec3 V, glm::vec3 W, glm::vec3 &P, glm::vec3 &N, VertexData* vertData, TriangleMesh* trimesh, int numOfMeshes)
{
	bool hit = false;
	float z = 1000.f;

	//loop through all triangles in the scene
	for (int k = 0; k < numOfMeshes; k++)
	{
		for (int i = 0; i < trimesh->numOfTris; i++)
		{
			//vertex indexes begin at 1 for obj files, thus the -1
			if (triangleTest(V, W, *(vertData->verts + ((trimesh + k)->tris + i)->x - 1), *(vertData->verts + ((trimesh + k)->tris + i)->y - 1),
				*(vertData->verts + ((trimesh + k)->tris + i)->z - 1), *(vertData->vnorms + ((trimesh + k)->tris + i)->vn1 - 1), P))
			{ //Did we hit something?
				//is this point closer to the camera? 
				float dist = glm::distance(V, P);
				if (dist < z)
				{
					N = *(vertData->vnorms + ((trimesh + k)->tris + i)->vn1 - 1);

					//we are closer so update the z position
					z = dist;
				}
				hit = true;
			}
		}
	}

	//shade the closest point if we hit something
	return hit;
}

/*Returns true if any triangle is hit*/
__device__ inline bool testTrianglesAny(glm::vec3 V, glm::vec3 W, VertexData* vertData, TriangleMesh* trimesh, int numOfMeshes)
{
	glm::vec3 P;

	//loop through all triangles in the scene
	for (int k = 0; k < numOfMeshes; k++)
	{
		for (int i = 0; i < trimesh->numOfTris; i++)
		{
			//vertex indexes begin at 1 for obj files, thus the -1
			if (triangleTest(V, W, *(vertData->verts + ((trimesh + k)->tris + i)->x - 1), *(vertData->verts + ((trimesh + k)->tris + i)->y - 1),
				*(vertData->verts + ((trimesh + k)->tris + i)->z - 1), *(vertData->vnorms + ((trimesh + k)->tris + i)->vn1 - 1), P))
			{ //Did we hit something?
				return true;
			}
		}
	}

	return false;
}

__device__ glm::vec3 shadePoint(glm::vec3 P, glm::vec3 W, glm::vec3 N, VertexData* vertData, TriangleMesh* trimesh, material* m, pointLight* lights, int num_lights, int numOfMeshes)
{
	glm::vec3 c = glm::vec3(0.f);
	for (int l = 0; l < num_lights; l++)
	{
		glm::vec3 contribution = glm::vec3(0.f);

		//calculate the light direction from the point lamps
		glm::vec3 Ld = (lights + l)->location - P;
		float distance = length(Ld);
		distance = distance * distance;
		Ld = normalize(Ld);

		// TODO: shadows for all surfaces

		//shadows from other triangles
		//trace from the point back to the light, is another triangle in the way?
		if (testTrianglesAny(P, Ld, vertData, trimesh, numOfMeshes))
		{
			continue;
		}

		glm::vec3 Rd = 2.f * N * dot(N, Ld) - Ld;
		contribution += (lights + l)->color * m->diffuse * glm::max(0.f, dot(N, Ld));
		contribution += m->specular * pow(glm::max(0.f, dot(Rd, -W)), m->power);
		contribution *= (lights + l)->i / distance;
		c += contribution;
	}

	return m->ambient + c;
	//return glm::vec3(1.f);
}

/*
__device__ inline glm::vec3 raySpheres(float tMin, glm::vec3 V, glm::vec3 W, sphere* spheres, int num_spheres, glm::vec3 color, material m, pointLight* lights, int num_lights)
{
	for (int n = 0; n < num_spheres; n++)
	{
		float t = raySphere(V, W, *(spheres + n));
		if (t > 0. && t < tMin) {
			glm::vec3 P = V + (W * t);
			glm::vec3 N = normalize(P - (spheres + n)->location);
			color = shadeSurface(P, W, N, spheres, num_spheres, m, lights, num_lights);
			tMin = t;
		}
	}
	return color;
}*/

struct Camera
{
	//set the film size
	const float image_plane_width = 0.07f; // 7cm
	const float image_plane_height = 0.06f; // 6cm

	//select a focal length, aperature, and focus distance
	const float focal_length = 0.05f; // 50mm
	const float aperture = 16.0f; // f16
	const float focus_dist = 3.f; // 3m
};

//Returns a point on a concentric disk, takes x and y values between -1 and 1
__device__ inline void sampleDisk(float &x, float &y)
{
	static const float PiOver2 = 1.570796326794896619231321691639;
	static const float PiOver4 = 0.785398163397448309615660845819;

	float theta, r;

	if (abs(x) > abs(y)) 
	{
		r = x;
		theta = PiOver4 * (y / x);
	}
	else 
	{
		r = y;
		theta = PiOver2 - PiOver4 * (x / y);
	}

	x = r * cos(theta);
	y = r * sin(theta);
}
/*
//Takes a ray, V, W, and returns a new ray direction using a lens sample point
__device__ inline void dofRay(glm::vec3 &V, glm::vec3 &W, const Camera& camera)
{
	//Sample a point on a disk, SP (sample point)
	float x, y, z;

	x = SimplexNoise::noise((float) blockDim.x * blockIdx.x + threadIdx.x);
	y = SimplexNoise::noise((float) gridDim.x * blockIdx.x + threadIdx.x);
	z = SimplexNoise::noise(gridDim.x + (float) blockIdx.x * threadIdx.x);

	if (z > -0.5f)
	{
		x = -x;
	}
	else if (z > 0.f)
	{
		y = -y;
	}
	else if (z > 0.5f)
	{
		x = -x;
		y = -y;
	}

	sampleDisk(x, y);

	//Convert unit disk coordinates to lens diameter, LP (lens point)
	V.x = x * camera.focal_length / camera.aperture;
	V.y = y * camera.focal_length / camera.aperture;

	//calculate the focal point by scaling the primary ray and adding it to the ray origin position
	glm::vec3 focalPoint = V + (W * camera.focus_dist);

	//Construct a unit vector from the sample point to the focal point, FD (focal direction)
	W = normalize(focalPoint - V);
}*/

__device__ inline glm::vec3 shadePoint(const Camera &camera, const float pixel_x, const float pixel_y, VertexData* vertData, TriangleMesh* trimesh, int numOfMeshes)
{
	//find this pixel's camera space position
	glm::vec3 pixelpos;
	pixelpos.x = pixel_x / blockDim.x;
	pixelpos.x = pixelpos.x * camera.image_plane_width - (camera.image_plane_width / 2.0f);
	pixelpos.y = pixel_y / gridDim.x;
	pixelpos.y = -pixelpos.y * camera.image_plane_height + (camera.image_plane_height / 2.0f);
	pixelpos.z = 0.f;

	//Set V and W ... TODO add lens configurations
	glm::vec3 V = glm::vec3(0.f, 0.f, camera.focal_length);
	glm::vec3 W = glm::vec3(V.x + pixelpos.x, V.y + pixelpos.y, -1.f * V.z);

	//dofRay(V, W, camera);

	//set the background color
	glm::vec3 color = glm::vec3(0.6f, 0.8f, 0.8f);

	// TEMP DEFINE LIGHT DATA
	const int num_lights = 2;
	pointLight lights[num_lights] = { pointLight(0.5f, 5.5f, -5.5f, .59f, .93f, .59f, 25.f), pointLight(1.5f, -.5f, -1.5f, .19f, .63f, .49f, 3.f) };
	// TEMP DEFINE MATERIAL DATA
	material sphere_mat = material(0.1f, 0.1f, 0.1f, 0.2f, 0.4f, 0.5f, 1.f, 1.f, 1.f, 1.5f);

	glm::vec3 P;
	glm::vec3 N;

	if (testTriangles(V, W, P, N, vertData, trimesh, numOfMeshes))
	{
		color = shadePoint(P, W, N, vertData, trimesh, &sphere_mat, &lights[0], num_lights, numOfMeshes);
	}

	return color;
}

__global__ void shadePixels(glm::vec3* pixelptr, VertexData* vertData, TriangleMesh* trimesh, int numOfMeshes)
{
	int pixel_x = threadIdx.x;
	int pixel_y = blockIdx.x;

	//pixels are stored colunm/block num by row/thread offset
	//-----------------------------
	//---------------x-------------
	//-----------------------------
	pixelptr = pixelptr + (pixel_y * blockDim.x) + pixel_x;

	Camera camera;

	const int num_pixel_samples = 4; //must be divisible by 2

	//hiprandState_t state;

	glm::vec3 color = glm::vec3(0.);

	for (int i = 0; i < num_pixel_samples; i++)
	{
		//TODO: divide the pixel further into cells
		/*for (int j = 0; j < num_pixel_samples; j++)
		{
			color += shadePoint(camera, pixel_x + (1 / num_pixel_samples) * i + (1 / (num_pixel_samples * 2)), 
			pixel_y + (1 / num_pixel_samples) * j + (1 / (num_pixel_samples * 2)), trimesh);
		}*/
		
		//this takes samples in an x pattern
		float rand_x, rand_y;
		if (i < num_pixel_samples/2)
		{
			rand_x = pixel_x + 1 / num_pixel_samples * i;
			rand_y = pixel_y + 1 / num_pixel_samples * i;
		}
		else
		{
			rand_x = (pixel_x + 1) - 1 / num_pixel_samples * i;
			rand_y = pixel_y + 1 / num_pixel_samples * i;
		}
		color += shadePoint(camera, rand_x,	rand_y, vertData, trimesh, numOfMeshes);

		//select a random point within the interval
		//hiprand_init(2127+i, (blockDim.x * pixel_y + pixel_x), 0, &state);
		//float rand_x = pixel_x + hiprand_uniform(&state) * pixel_width;
		//float rand_y = pixel_y + hiprand_uniform(&state) * pixel_height;
	}

	//average the pixel sample colors
	color /= num_pixel_samples;

	//Set the final color
	pixelptr->r = glm::clamp(color.x * 255.f, 0.f, 255.f);
	pixelptr->g = glm::clamp(color.y * 255.f, 0.f, 255.f);
	pixelptr->b = glm::clamp(color.z * 255.f, 0.f, 255.f);

}

int main()
{
	const int image_width = 1024;
	const int image_height = 878;

	//create the pixel array
	glm::vec3* pixels;

	hipMallocManaged(&pixels, sizeof(glm::vec3) * image_width * image_height);

	
	//Read in Triangle information from OBJ file
	TriangleMesh* trimeshes;
	VertexData* vertData;
	const int maxMeshes = 3;
	const int maxVertices = 300; //For the entire scene
	const int maxFaces = 300; //Per object
	
	hipMallocManaged(&trimeshes, sizeof(TriangleMesh) * maxMeshes);
	hipMallocManaged(&vertData, sizeof(vertData));

	glm::vec3* verts;
	glm::vec3* vnorms;
	hipMallocManaged(&verts, sizeof(glm::vec3) * maxVertices * maxMeshes);
	hipMallocManaged(&vnorms, sizeof(glm::vec3) * maxVertices * maxMeshes);
	vertData->verts = verts;
	vertData->vnorms = vnorms;

	//Initialize the triangle arrays for all meshes
	for (int i = 0; i < maxMeshes; i++)
	{
		face* tris;
		
		hipMallocManaged(&tris, sizeof(face) * maxFaces);
		
		(trimeshes + i)->tris = tris;
	}
	

	auto start_p0 = std::chrono::system_clock::now();

	int numOfMeshes; //we don't know how many there will be so save the result here
	parseOBJFile("in/mnky.obj", vertData, trimeshes, maxMeshes, numOfMeshes);

	auto start_p1 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p0 = start_p0 - start_p1;

	//run the kernel, <<< numOf Blocks, numOf threads per block >>>
	shadePixels <<<image_height, image_width>>> (pixels, vertData, trimeshes, numOfMeshes);

	hipDeviceSynchronize();

	auto start_p2 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p1 = start_p2 - start_p1;

	//save the output to a file
	std::ofstream image_of;
	image_of.open("out/img.ppm", std::ios::out);

	if (!image_of.is_open())
	{
		std::cout << "File does not exist";
	}

	image_of << "P3\n" << image_width << ' ' << image_height << "\n255\n";

	for (int i = 0; i < image_height; i++)
	{ 
		for (int k = 0; k < image_width; k++)
		{
			glm::vec3* p = pixels + (i * image_width) + k;
			
			image_of << static_cast<int>(p->r) << ' ' << static_cast<int>(p->g) << ' ' << static_cast<int>(p->b) << ' ';
		}
		image_of << "\n";
	}

	auto start_p3 = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_p2 = start_p3 - start_p2;

	std::cout << "Obj input elapsed time: " << elapsed_p0.count() << "s\n";
	std::cout << "GPU computation elapsed time: " << elapsed_p1.count() << "s\n";
	std::cout << "Image output elapsed time: " << elapsed_p2.count() << "s\n";

	//clean up
	image_of.close();

	for (int i = 0; i < maxMeshes; i++)
	{
		
		hipFree((trimeshes + i)->tris);
	}
	hipFree(trimeshes);

	hipFree(vertData->verts);
	hipFree(vertData->vnorms);
	hipFree(vertData);
	
	hipFree(pixels);
	hipDeviceReset();

	return 0;
}